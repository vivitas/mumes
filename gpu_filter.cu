#include "gpu_filter.h"
#include "common_filter.h"
#include "ScopeClock.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void
gpu_filter
(
        Array2D<Rgba> &pixels,
        int width,
        int height,
        t_times &time
)
{
    t_my_rgba *input_buffer, *output_buffer;
    alloc_buffers(input_buffer, output_buffer, width, height);
    t_my_rgba *device_input;
    
    HANDLE_ERROR(hipMalloc((void**)&device_input, sizeof(t_my_rgba)*width*height));
    {
        ScopeClock dummy(&time.transfer_to);
        copy(input_buffer, pixels, width, height);
    }
    {
        ScopeClock dummy(&time.processing);
    }
    {
        ScopeClock dummy(&time.transfer_from);
    }
    hipFree(device_input);
    free_buffers(input_buffer, output_buffer);
}
