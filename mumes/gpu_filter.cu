#include "hip/hip_runtime.h"
#include "gpu_filter.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ScopeTimer.h"

#define TPB 64

static char error[1024];
static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if(err != hipSuccess)
    {        
        sprintf(error, "%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        throw error;
    }
}
#define MAX(A, B) (A>B?A:B)
#define MIN(A, B) (A<B?A:B)
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
__global__ void
gpu_kernel
(
    t_RGBA *src,
    t_RGBA *dst,
    int width,
    int height,
    int depth
)
{
    int real_x = blockIdx.x;
    int real_y = blockIdx.y * TPB + threadIdx.x;
    if(real_x >= width || real_y >= height)
        return;
    int x_from = MAX(real_x - 1, 0);
    int y_from = MAX(real_y - 1, 0);
    int x_to = MIN(real_x + 1, width - 1);
    int y_to = MIN(real_y + 1, height - 1);
    int coef = (x_to - x_from + 1) * (y_to - y_from - 1);
 
    dst[real_x + width * real_y].r = 0.f;
    dst[real_x + width * real_y].g = 0.f;
    dst[real_x + width * real_y].b = 0.f;
    dst[real_x + width * real_y].a = 0.f;
        
    for(int x_ = x_from; x_ <= x_to; ++x_)
    {
        for(int y_ = y_from; y_ <= y_to; ++y_)
        {            
            dst[real_x + width * real_y].r += src[x_ + width*y_].r / coef;
            dst[real_x + width * real_y].g += src[x_ + width*y_].g / coef;
            dst[real_x + width * real_y].b += src[x_ + width*y_].b / coef;
            dst[real_x + width * real_y].a += src[x_ + width*y_].a / coef;
        }
    }
    //dst[blockIdx.x + width * blockIdx.y] = b;
}
t_timing
gpu_filter
(
    t_RGBA *&raw,
    int width,
    int height,
    int depth,
    int repetitions
)
{
    t_timing result;
    t_RGBA *gpu_src, *gpu_dst;
    {
        ScopeTimer _dummy(&(result.utilities_time));
        HANDLE_ERROR(hipMalloc(&gpu_src, width*height*depth*sizeof(t_RGBA)));
        HANDLE_ERROR(hipMalloc(&gpu_dst, width*height*depth*sizeof(t_RGBA)));
    }
    try
    {
        {
            ScopeTimer _dummy(&(result.transfer_time));
            HANDLE_ERROR(hipMemcpy(gpu_src, raw, width*height*depth*sizeof(t_RGBA), hipMemcpyHostToDevice));
        }
        {
            
            dim3 grid(width, height/TPB);
            for(int i = 0; i < repetitions; ++i)
            {
                {
                    ScopeTimer _dummy(&(result.processing_time));
                    gpu_kernel << < grid, TPB >> > (gpu_src, gpu_dst, width, height, depth);
                    HANDLE_ERROR(hipGetLastError());
                
                    t_RGBA *tmp;
                    tmp = gpu_src;
                    gpu_src = gpu_dst;
                    gpu_dst = tmp;
                }
            }        
        }
        {
            ScopeTimer _dummy(&(result.transfer_time));
            HANDLE_ERROR(hipMemcpy(raw, gpu_src, width*height*depth*sizeof(t_RGBA), hipMemcpyDeviceToHost));
        }
        {
            ScopeTimer _dummy(&(result.utilities_time));
            HANDLE_ERROR(hipFree(gpu_src));
            HANDLE_ERROR(hipFree(gpu_dst));
        }
    }
    catch(char* c)
    {
        ScopeTimer _dummy(&(result.utilities_time));
        try
        {
            HANDLE_ERROR(hipFree(gpu_src));
        }
        catch(...)
        {
        }
        try
        {
            HANDLE_ERROR(hipFree(gpu_dst));
        }
        catch(...)
        {
        }
        throw c;
    }
    return result;
}
void
prepare_cuda_device
(
)
{
    for(int i=0; i<10; ++i)
    {
        void* testMemory;
        HANDLE_ERROR(hipMalloc(&testMemory, 1024 * 768 * 4 * 4));
        HANDLE_ERROR(hipFree(testMemory));
    }
}