#include "hip/hip_runtime.h"
#include "gpu_filter.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        getchar();
        getchar();
        exit(EXIT_FAILURE);
    }
}
#define MAX(A, B) (A>B?A:B)
#define MIN(A, B) (A<B?A:B)
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
__global__ void
gpu_kernel
(
    float *src,
    float *dst,
    int width,
    int height,
    int depth
)
{
    if(blockIdx.x >= width || blockIdx.y >= height || threadIdx.x >= 4)
        return;
    int x_from = MAX(blockIdx.x - 1, 0);
    int y_from = MAX(blockIdx.y - 1, 0);
    int x_to = MIN(blockIdx.x + 1, width - 1);
    int y_to = MIN(blockIdx.y + 1, height - 1);
    int coef = (x_to - x_from + 1) * (y_to - y_from - 1);
    dst[threadIdx.x + 4 * blockIdx.x + 4 * width*blockIdx.y] = 0.f;
 
    float b = 0;
    
    for(int x_ = x_from; x_ <= x_to; ++x_)
    {
        for(int y_ = y_from; y_ <= y_to; ++y_)
        {            
            b += src[threadIdx.x + 4 * x_ + 4 * width*y_] / coef;
        }
    }
    dst[threadIdx.x + 4 * blockIdx.x + 4 * width * blockIdx.y] = b;
}
void
gpu_filter
(
    t_RGBA *raw,
    int width,
    int height,
    int depth
)
{
    float *gpu_src, *gpu_dst;
    HANDLE_ERROR(hipMalloc(&gpu_src, width*height*depth*sizeof(t_RGBA)));
    HANDLE_ERROR(hipMalloc(&gpu_dst, width*height*depth*sizeof(t_RGBA)));
    HANDLE_ERROR(hipMemcpy(gpu_src, raw, width*height*depth*sizeof(t_RGBA), hipMemcpyHostToDevice));
    dim3 grid(width, height);
    gpu_kernel <<< grid, 4 >> > (gpu_src, gpu_dst, width, height, depth);
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipMemcpy(raw, gpu_dst, width*height*depth*sizeof(t_RGBA), hipMemcpyDeviceToHost));
    hipFree(gpu_src);
    hipFree(gpu_dst);
}