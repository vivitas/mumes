#include "hip/hip_runtime.h"
#include "gpu_filter.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ScopeTimer.h"
static char error[1024];
static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if(err != hipSuccess)
    {        
        sprintf(error, "%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        throw error;
    }
}
#define MAX(A, B) (A>B?A:B)
#define MIN(A, B) (A<B?A:B)
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
__global__ void
gpu_kernel
(
    float *src,
    float *dst,
    int width,
    int height,
    int depth
)
{
    if(blockIdx.x >= width || blockIdx.y >= height || threadIdx.x >= 4)
        return;
    int x_from = MAX(blockIdx.x - 1, 0);
    int y_from = MAX(blockIdx.y - 1, 0);
    int x_to = MIN(blockIdx.x + 1, width - 1);
    int y_to = MIN(blockIdx.y + 1, height - 1);
    int coef = (x_to - x_from + 1) * (y_to - y_from - 1);
    dst[threadIdx.x + 4 * blockIdx.x + 4 * width*blockIdx.y] = 0.f;
 
    float b = 0;
    
    for(int x_ = x_from; x_ <= x_to; ++x_)
    {
        for(int y_ = y_from; y_ <= y_to; ++y_)
        {            
            b += src[threadIdx.x + 4 * x_ + 4 * width*y_] / coef;
        }
    }
    dst[threadIdx.x + 4 * blockIdx.x + 4 * width * blockIdx.y] = b;
}

t_timing
gpu_filter
(
    t_RGBA *&raw,
    int width,
    int height,
    int depth,
    int repetitions
)
{
    t_timing result;
    float *gpu_src, *gpu_dst;
    {
        ScopeTimer _dummy(&(result.utilities_time));
        HANDLE_ERROR(hipMalloc(&gpu_src, width*height*depth*sizeof(t_RGBA)));
        HANDLE_ERROR(hipMalloc(&gpu_dst, width*height*depth*sizeof(t_RGBA)));
    }
    {
        ScopeTimer _dummy(&(result.transfer_time));
        HANDLE_ERROR(hipMemcpy(gpu_src, raw, width*height*depth*sizeof(t_RGBA), hipMemcpyHostToDevice));
    }
    {
        ScopeTimer _dummy(&(result.processing_time));
        dim3 grid(width, height);
        for(int i = 0; i < repetitions; ++i)
        {
            gpu_kernel <<< grid, 4 >>> (gpu_src, gpu_dst, width, height, depth);
            float* tmp = gpu_dst;
            gpu_dst = gpu_src;
            gpu_src = tmp;
        }        
        HANDLE_ERROR(hipGetLastError());
    }
    {
        ScopeTimer _dummy(&(result.transfer_time));
        HANDLE_ERROR(hipMemcpy(raw, gpu_src, width*height*depth*sizeof(t_RGBA), hipMemcpyDeviceToHost));
    }
    {
        ScopeTimer _dummy(&(result.utilities_time));
        hipFree(gpu_src);
        hipFree(gpu_dst);
    }
    return result;
}